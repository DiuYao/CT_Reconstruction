#include "hip/hip_runtime.h"
#include "Kernel.cuh"

#include <stdio.h>


void reconGPU(ImagingSystemInfo mImagingSystemInfo, ReconInfoData h_mReconInfoData, ReconInfoData d_mReconInfoData, GeometryPara mGeometryPara)
{
	//chooseGPU(0);
	hipError_t cudaStatus;
	
	computeDetectorPoints(mImagingSystemInfo, d_mReconInfoData);

	filterFT(mImagingSystemInfo, h_mReconInfoData, d_mReconInfoData);

#if 0
	cudaStatus = hipMemcpy(h_mReconInfoData.filter, d_mReconInfoData.filter, mImagingSystemInfo.dNumU * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable h_mReconInfoData.filter hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
		exit(0);
	}

	for (int i = 0; i < mImagingSystemInfo.dNumU; i++)
	{
		std::cout << h_mReconInfoData.filter[i].x << " " << std::endl;
	}
#endif

	prepareReconVariables(mImagingSystemInfo, d_mReconInfoData);

#if 0
	cudaStatus = hipMemcpy(d_mReconInfoData.detTempV, d_mReconInfoData.x, mImagingSystemInfo.pNumX * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable h_mReconInfoData.detTempV hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
		exit(0);
	}

	for (int i = 0; i < mImagingSystemInfo.dNumV; i++)
	{
		std::cout << d_mReconInfoData.detTempV[i] << " " << std::endl;
	}
#endif


	dim3 blockSizeRec(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeRec((mImagingSystemInfo.pNumX - 1) / blockSizeRec.x + 1, (mImagingSystemInfo.pNumY - 1) / blockSizeRec.y + 1, (mImagingSystemInfo.pNumZ - 1) / blockSizeRec.z + 1);

	dim3 blockSizeWProj(BLOCKSIZEX, BLOCKSIZEY);
	dim3 gridSizeWProj((mImagingSystemInfo.dNumU - 1) / blockSizeWProj.x + 1, (mImagingSystemInfo.dNumV - 1) / blockSizeWProj.y + 1);

	// ����ͶӰ����FFT hipfftPlan1d
	hipfftHandle planProj;                           // Create cuda library function handle
	hipfftPlan1d(&planProj, mImagingSystemInfo.dNumU, HIPFFT_C2C, mImagingSystemInfo.dNumV);    // Plan declaration

	// �����ؽ������2D����ͶӰ����
	hipArray_t cuArray;
	hipTextureObject_t texObj = 0;
	createTexture2D(texObj, cuArray, d_mReconInfoData.filterProj, mImagingSystemInfo.dNumU, mImagingSystemInfo.dNumV);


	// ----------------------------��Ƕ��ؽ�
	std::cout << "�ؽ���ʼ ..." << std::endl;
	float angle = 0.0f;
	// Timing


	std::cout << "����� ";
	HANDLE hConsole;
	COORD coord;
	getCurrentCursorCoordinate(hConsole, coord);

	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	for (size_t i = 0; i < mImagingSystemInfo.views; i++)        
	{
		//std::cout << "�� " << i + 1 << " ���Ƕ�" << "==>>" << std::endl;

		// ��ȡһ���Ƕȵ�ͶӰ
		for (size_t j = 0; j < mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV; j++)
		{
			h_mReconInfoData.proj[j] = h_mReconInfoData.totalProj[j + i * mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV];
		}


		cudaStatus = hipMemcpy(d_mReconInfoData.proj, h_mReconInfoData.proj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Variable d_mReconInfoData.proj hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
#if 0
		for (size_t j = 0; j < mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV; j++)
		{
			std::cout << h_mReconInfoData.proj[j] << " ";
		}
#endif
		// 1. weight the projection
		weightProjection << <gridSizeWProj, blockSizeWProj >> > (mImagingSystemInfo, d_mReconInfoData);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "weightKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching weightKernel!\n", cudaStatus);
			exit(0);
		}

#if 0
		cudaStatus = hipMemcpy(h_mReconInfoData.weightProj, d_mReconInfoData.weightProj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Variable d_mReconInfoData.proj hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		for (size_t j = 0; j < mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV; j++)
		{
			std::cout << h_mReconInfoData.weightProj[j].x << " ";
		}
#endif


		// 2. Filter projections in the frequency domain
		// ͶӰ����FFT
		//hipMalloc(&d_mReconInfoData.wfProj, )
		hipfftExecC2C(planProj, (hipfftComplex*)d_mReconInfoData.weightProj, (hipfftComplex*)d_mReconInfoData.weightProj, HIPFFT_FORWARD);  //execute FFT

#if 0
		cudaStatus = hipMemcpy(h_mReconInfoData.weightProj, d_mReconInfoData.weightProj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Variable h_mReconInfoData.filter hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		for (int j = 0; j < mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV; j++)
		{
			std::cout << h_mReconInfoData.weightProj[j].x << " + " << h_mReconInfoData.weightProj[j].y << " i" << "  ";
		}
#endif

		// Ƶ����ͶӰ�����˲�
		projectionFilterInTDF << <gridSizeWProj, blockSizeWProj >> > (mImagingSystemInfo, d_mReconInfoData);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "projectionFilterInTDFKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching projectionFilterInTDFKernel!\n", cudaStatus);
			exit(0);
		}

#if 0
		cudaStatus = hipMemcpy(h_mReconInfoData.weightProj, d_mReconInfoData.weightProj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Variable h_mReconInfoData.filter hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		for (int j = 0; j < mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV; j++)
		{
			std::cout << h_mReconInfoData.weightProj[j].x << " + " << h_mReconInfoData.weightProj[j].y << " i" << "  ";
		}
#endif

		// ͶӰ����IFFT
		hipfftExecC2C(planProj, (hipfftComplex*)d_mReconInfoData.weightProj, (hipfftComplex*)d_mReconInfoData.weightProj, HIPFFT_BACKWARD);    // execute IFFT
		processFilteredProjIfft << <gridSizeWProj, blockSizeWProj >> > (mImagingSystemInfo, d_mReconInfoData);        // ���Ա任�����еĸ���

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "processFilteredProjIfftKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching processFilteredProjIfftKernel!\n", cudaStatus);
			exit(0);
		}

		// ����˲����ͶӰ
		cudaStatus = hipMemcpy(h_mReconInfoData.filterProj, d_mReconInfoData.filterProj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Variable d_mReconInfoData.proj hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		/*for (size_t j = 0; j < mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV; j++)
		{
			std::cout << h_mReconInfoData.filterProj[j] << " ";
		}*/

		for (size_t j = 0; j < mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV; j++)
		{
			h_mReconInfoData.totalProj[mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * i + j] = h_mReconInfoData.filterProj[j];
		}

		// ----------------------------------��Ͷ------------------------------------
		// ��������
		hipMemcpyToArray(cuArray, 0, 0, d_mReconInfoData.filterProj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(float), hipMemcpyDeviceToDevice);  // ��������

		angle = i * mImagingSystemInfo.thetaStep * mImagingSystemInfo.RotatedDirection;

		reconstructeImage << <gridSizeRec, blockSizeRec >> > (angle, texObj, mImagingSystemInfo, d_mReconInfoData, mGeometryPara);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reconstructeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reconstructeKernel!\n", cudaStatus);
		}

		SetConsoleCursorPosition(hConsole, coord);  //�ƶ���� 
		/*std::cout.precision(2);
		std::cout << (float)(i+1) / mImagingSystemInfo.views * 100 << "%" << std::endl;*/
		printf("%.2f%%\n", (float)(i + 1) / mImagingSystemInfo.views * 100);
	
	}
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>�ؽ�ͼ������ʱ��(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);

	size_t sizeImage = mImagingSystemInfo.pNumX * mImagingSystemInfo.pNumY * mImagingSystemInfo.pNumZ * sizeof(float);
	cudaStatus = hipMemcpy(h_mReconInfoData.imageRecon, d_mReconInfoData.imageRecon, sizeImage, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "variable h_mReconInfoData.imageRecon hipMemcpy failed!");
	}
	std::cout << "�ؽ���ɣ�" << std::endl;
	



	CUDAFREE(d_mReconInfoData.detU);
	CUDAFREE(d_mReconInfoData.detV);
	CUDAFREE(d_mReconInfoData.filter);
	CUDAFREE(d_mReconInfoData.filterProj);
	CUDAFREE(d_mReconInfoData.imageRecon);
	CUDAFREE(d_mReconInfoData.proj);
	CUDAFREE(d_mReconInfoData.totalProj);
	CUDAFREE(d_mReconInfoData.weightProj);
	CUDAFREE(d_mReconInfoData.x);
	CUDAFREE(d_mReconInfoData.y);
	CUDAFREE(d_mReconInfoData.z);
}


// Choose which GPU to run on, change this on a multi-GPU system.
void chooseGPU(uchar index)
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(index);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}



void computeDetectorPoints(ImagingSystemInfo mImagingSystemInfo, ReconInfoData& d_mReconInfoData)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc(&d_mReconInfoData.detU, mImagingSystemInfo.dNumU * sizeof(float));
	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mReconInfoData.detU hipMalloc failed!");
	}

	cudaStatus = hipMalloc(&d_mReconInfoData.detV, mImagingSystemInfo.dNumV * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mReconInfoData.detV hipMemcpy failed!");
	}

	dim3 blockSizeIPC(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mImagingSystemInfo.dNumU - 1) / blockSizeIPC.x + 1, (mImagingSystemInfo.dNumV - 1) / blockSizeIPC.y + 1);

	// Timing
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	computeDetecPointsCoorsKernel << <gridSizeIPC, blockSizeIPC >> > (mImagingSystemInfo, d_mReconInfoData);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n", cudaStatus);
	}

	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0.0f;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);

	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);

	//totalTime += elapsedTime;

	/*hipMemcpy(d_mReconInfoData.detV, d_mReconInfoData.detU, mImagingSystemInfo.dNumU * sizeof(float), hipMemcpyDeviceToHost);*/
#if 0
	hipMemcpy(d_mReconInfoData.detV, d_mReconInfoData.detV, mImagingSystemInfo.dNumV * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < mImagingSystemInfo.dNumV; i++)
	{
		std::cout << d_mReconInfoData.detV[i] << " " << std::endl;
	}

#endif
	std::cout << "����̽������Ԫ��������ʱ��(GPU)��" << elapsedTime << " ms" << std::endl;
}

void filterFT(ImagingSystemInfo mImagingSystemInfo, ReconInfoData& h_mReconInfoData, ReconInfoData& d_mReconInfoData)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc(&d_mReconInfoData.filter, mImagingSystemInfo.dNumU * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mReconInfoData.filter hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(d_mReconInfoData.filter, h_mReconInfoData.filter, mImagingSystemInfo.dNumU * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "variable d_mReconInfoData.filter hipMemcpy failed!");
	}
	// �˲���FFT
	hipfftHandle planFilter;                           // Create cuda library function handle
	hipfftPlan1d(&planFilter, mImagingSystemInfo.dNumU, HIPFFT_C2C, 1);    // Plan declaration
	hipfftExecC2C(planFilter, (hipfftComplex*)d_mReconInfoData.filter, (hipfftComplex*)d_mReconInfoData.filter, HIPFFT_FORWARD);  //execute FFT
	// ȡģ
	filterAmplitude << <mImagingSystemInfo.dNumU / 1024 + 1, 1024 >> > (d_mReconInfoData.filter, mImagingSystemInfo.dNumU);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "filterAmplitudeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching filterAmplitudeKernel!\n", cudaStatus);
	}

#if 0
	cudaStatus = hipMemcpy(h_mReconInfoData.filter, d_mReconInfoData.filter, mImagingSystemInfo.dNumU * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable h_mReconInfoData.filter hipMemcpy failed! %s.\n", hipGetErrorString(cudaStatus));
		exit(0);
	}

	/*for (int i = 0; i < mImagingSystemInfo.dNumU; i++)
	{
		std::cout << h_mReconInfoData.filter[i].x << " " << std::endl;
	}*/
#endif // 0
}

void prepareReconVariables(ImagingSystemInfo mImagingSystemInfo, ReconInfoData& d_mReconInfoData)
{
	hipError_t cudaStatus;

	// �洢�ؽ�ͼ�������������
	cudaStatus = hipMalloc(&d_mReconInfoData.x, mImagingSystemInfo.pNumX * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed!");
	}
	cudaStatus = hipMalloc(&d_mReconInfoData.y, mImagingSystemInfo.pNumY * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed!");
	}
	cudaStatus = hipMalloc(&d_mReconInfoData.z, mImagingSystemInfo.pNumZ * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed!");
	}

	dim3 blockSizeRec(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeRec((mImagingSystemInfo.pNumX - 1) / blockSizeRec.x + 1, (mImagingSystemInfo.pNumY - 1) / blockSizeRec.y + 1, (mImagingSystemInfo.pNumZ - 1) / blockSizeRec.z + 1);

	// �����ؽ�ͼ������
	computeImgCoordinates << <gridSizeRec, blockSizeRec >> > (mImagingSystemInfo, d_mReconInfoData);
	
	//hipStreamSynchronize(0);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeImgCoordinatesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		exit(0);
	}

	// ��ȡÿ���Ƕ�ͶӰ�������
	cudaStatus = hipMalloc(&d_mReconInfoData.proj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mReconInfoData.proj hipMalloc failed!");
	}

	// ��Ȩ����ͶӰ
	cudaStatus = hipMalloc(&d_mReconInfoData.weightProj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mReconInfoData.weightProj hipMalloc failed!");
	}

	// �˲��������
	cudaStatus = hipMalloc(&d_mReconInfoData.filterProj, mImagingSystemInfo.dNumU * mImagingSystemInfo.dNumV * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mReconInfoData.filterProj hipMalloc failed!");
	}

	// ��ͶӰ�������
	cudaStatus = hipMalloc(&d_mReconInfoData.imageRecon, mImagingSystemInfo.pNumX * mImagingSystemInfo.pNumY * mImagingSystemInfo.pNumZ * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mReconInfoData.imageRecon hipMalloc failed!");
	}

}

// ������ά�������ݴ�����Device to Device��
// texObj -- �������cuArray -- Device�д洢���ݵ�ָ�룬data -- Դ����
void createTexture2D(hipTextureObject_t& texObj, hipArray_t& cuArray, float* data, size_t width, size_t height)
{
	/* ���� hipTextureObject_t ���� */
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	//hipArray_t cuArray;
	hipMallocArray(&cuArray, &channelDesc, width, height);

	// Set pitch of the source (the width in memory in bytes of the 2D array pointed to by src,
	// including paddding), we dont have any padding

	// Copy data located at address h_data in host memory to device memory
	//hipMemcpy2DToArray(cuArray, 0, 0, h_data, spitch, width * sizeof(float), height, hipMemcpyHostToDevice);  // �⴮����ᵼ�º���memset()����.
	hipMemcpyToArray(cuArray, 0, 0, data, width * height * sizeof(float), hipMemcpyDeviceToDevice);

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	// Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;   // Խ����䷽ʽ
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;                     // ����ԭʼ�������ͣ�����һ��

	// Create texture object
	//hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
}

// HANDLE hConsole ���� ��Ļ�ߴ� ������COORD coord ���� ������� ����  ���x��  ���y�� 
void getCurrentCursorCoordinate(HANDLE& hConsole, COORD& coord)
{
	hConsole = GetStdHandle(STD_OUTPUT_HANDLE);  //�����Ļ�ߴ� 
	CONSOLE_SCREEN_BUFFER_INFO csbi;
	GetConsoleScreenBufferInfo(hConsole, &csbi);
	short x = csbi.dwCursorPosition.X;
	short y = csbi.dwCursorPosition.Y;
	coord = { x,  y };
}




// ----------------------------------- Kernel function --------------------------------------------

__global__ void computeDetecPointsCoorsKernel(ImagingSystemInfo mImagingSystemInfo, ReconInfoData d_mReconInfoData)
{
	// ������ֵ�����
	unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int z = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < mImagingSystemInfo.dNumU && z < mImagingSystemInfo.dNumV)
	{
		// ̽������Ԫ����
		d_mReconInfoData.detU[y] = -mImagingSystemInfo.dHalfLU + mImagingSystemInfo.dSize / 2 + y * mImagingSystemInfo.dSize;
		d_mReconInfoData.detV[z] = -mImagingSystemInfo.dHalfLV + mImagingSystemInfo.dSize / 2 + z * mImagingSystemInfo.dSize;
	}
}

__global__ void filterAmplitude(hipfftComplex* d_filter, size_t dNum)
{
	size_t x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x < dNum)
	{
		d_filter[x].x = sqrtf(powf(d_filter[x].x, 2) + powf(d_filter[x].y, 2));  // ��ʵ���洢ģ
	}
}

__global__ void computeImgCoordinates(ImagingSystemInfo mImagingSystemInfo, ReconInfoData d_mReconInfoData)
{
	size_t x = threadIdx.x + blockIdx.x * blockDim.x;
	size_t y = threadIdx.y + blockIdx.y * blockDim.y;
	size_t z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x < mImagingSystemInfo.pNumX) //&& y < mImagingSystemInfo.pNumY && z < mImagingSystemInfo.pNumZ)
	{
		// �˴�������������ϵΪ��׼�� �����ؽ�ͼ�����������
		d_mReconInfoData.x[x] = -mImagingSystemInfo.imgReconLenX / 2 + mImagingSystemInfo.pSizeX / 2.0 + x * mImagingSystemInfo.pSizeX;
		d_mReconInfoData.y[y] = -mImagingSystemInfo.imgReconLenY / 2 + mImagingSystemInfo.pSizeY / 2.0 + y * mImagingSystemInfo.pSizeY;
		d_mReconInfoData.z[z] = -mImagingSystemInfo.imgReconLenZ / 2 + mImagingSystemInfo.pSizeZ / 2.0 + z * mImagingSystemInfo.pSizeZ;
	}
}

__global__ void weightProjection(ImagingSystemInfo mImagingSystemInfo, ReconInfoData d_mReconInfoData)
{
	size_t y = blockIdx.x * blockDim.x + threadIdx.x;
	size_t z = blockIdx.y * blockDim.y + threadIdx.y;
	if (y < mImagingSystemInfo.dNumU && z < mImagingSystemInfo.dNumV)
	{
		d_mReconInfoData.weightProj[z * mImagingSystemInfo.dNumU + y].x = d_mReconInfoData.proj[z * mImagingSystemInfo.dNumU + y] * mImagingSystemInfo.sdd / sqrtf(powf(d_mReconInfoData.detU[y], 2) + powf(d_mReconInfoData.detV[z], 2) + powf(mImagingSystemInfo.sdd, 2));
		d_mReconInfoData.weightProj[z * mImagingSystemInfo.dNumU + y].y = 0;
	}
}

__global__ void projectionFilterInTDF(ImagingSystemInfo mImagingSystemInfo, ReconInfoData d_mReconInfoData)
{
	size_t y = threadIdx.x + blockIdx.x * blockDim.x;
	size_t z = threadIdx.y + blockIdx.y * blockDim.y;
	if (y < mImagingSystemInfo.dNumU && z < mImagingSystemInfo.dNumV)
	{
		d_mReconInfoData.weightProj[z * mImagingSystemInfo.dNumU + y].x = d_mReconInfoData.weightProj[z * mImagingSystemInfo.dNumU + y].x * d_mReconInfoData.filter[y].x;
		d_mReconInfoData.weightProj[z * mImagingSystemInfo.dNumU + y].y = d_mReconInfoData.weightProj[z * mImagingSystemInfo.dNumU + y].y * d_mReconInfoData.filter[y].x;
	}
}

__global__ void processFilteredProjIfft(ImagingSystemInfo mImagingSystemInfo, ReconInfoData d_mReconInfoData)
{
	size_t x = threadIdx.x + blockIdx.x * blockDim.x;
	size_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < mImagingSystemInfo.dNumU && y < mImagingSystemInfo.dNumV)
	{
		d_mReconInfoData.filterProj[y * mImagingSystemInfo.dNumU + x] = d_mReconInfoData.weightProj[y * mImagingSystemInfo.dNumU + x].x / mImagingSystemInfo.dNumU;
	}
}

__global__ void reconstructeImage(float angle, hipTextureObject_t texProj, ImagingSystemInfo mImagingSystemInfo, ReconInfoData d_mReconInfoData, GeometryPara mGeometryPara)
{
	size_t x = threadIdx.x + blockIdx.x * blockDim.x;
	size_t y = threadIdx.y + blockIdx.y * blockDim.y;
	size_t z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x < mImagingSystemInfo.pNumX && y < mImagingSystemInfo.pNumY && z < mImagingSystemInfo.pNumZ)
	{
		angle = angle / 180 * PI;
		float tImgX = d_mReconInfoData.x[x] * cosf(angle) - d_mReconInfoData.y[y] * sinf(angle);
		float tImgY = d_mReconInfoData.y[x] * sinf(angle) + d_mReconInfoData.y[y] * cosf(angle);

		// ���ص������Ӧ��̽�����ϵ�����
		float u = mImagingSystemInfo.sdd * tImgY / (mImagingSystemInfo.sod + tImgX); //+ dHalfY) / dSize;
		float v = mImagingSystemInfo.sdd * d_mReconInfoData.z[z] / (mImagingSystemInfo.sod + tImgX); //+ dHalfZ) / dSize;
		
		// �˴��������̽����������������ϵΪ��׼�ģ�Ҳ��Ϊ�����嶥�����϶ˡ� ��ʱ��תУ��ʱ�����Ƕ�����ʱ�롣
		float correctedU = (u * cosf(mGeometryPara.beta) - v * sinf(mGeometryPara.beta) + mImagingSystemInfo.dHalfLU) / mImagingSystemInfo.dSize + mGeometryPara.offSetDetecW;    // offsetWָ̽������U(X)����
		float correctedV = (u * sinf(mGeometryPara.beta) + v * cosf(mGeometryPara.beta) + mImagingSystemInfo.dHalfLV) / mImagingSystemInfo.dSize + mGeometryPara.offSetDetecH;

		//imgRec[z * width * height + y * width + x] = v / pSizeY;

		//// Read from texture and write to global memory
		d_mReconInfoData.imageRecon[z * mImagingSystemInfo.pNumX * mImagingSystemInfo.pNumY + y * mImagingSystemInfo.pNumX + x] += (mImagingSystemInfo.sod * mImagingSystemInfo.sod) / ((mImagingSystemInfo.sod + tImgX) * (mImagingSystemInfo.sod + tImgX)) * tex2D<float>(texProj, correctedU, correctedV) * mImagingSystemInfo.thetaStep;// / mImagingSystemInfo.imgReconLenX; // (mImagingSystemInfo.pNumX * mImagingSystemInfo.pSizeX);
		//d_mReconInfoData.imageRecon[z * mImagingSystemInfo.pNumX * mImagingSystemInfo.pNumY + y * mImagingSystemInfo.pNumX + x] += 1 / 2.0f * (mImagingSystemInfo.sod * mImagingSystemInfo.sod) / ((mImagingSystemInfo.sod + tImgX) * (mImagingSystemInfo.sod + tImgX)) * tex2D<float>(texProj, correctedU + 0.5, correctedV + 0.5) * mImagingSystemInfo.thetaStep;// / mImagingSystemInfo.imgReconLenX; // (mImagingSystemInfo.pNumX * mImagingSystemInfo.pSizeX);
		
																																																																																					   // ���������ֵʱ���Ƿ���Ҫ��0.5   ????		
		/*imgRec[z * width * height + y * width + x] += ((sod * sdd) / pow((sod + imgX[x] * sinf(angle) - imgY[y] * cosf(angle)), 2))
			* tex3D<float>(texProj, u, v, num + 0.5) / width;*/
			// u, vӦ�ó���dSize��ĿǰdSize��1, ������Ҫ���Գ���dSize
	}
}
 